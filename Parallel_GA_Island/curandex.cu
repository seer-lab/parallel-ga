#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>

#define MAXTHREADS 2
#define NBBLOCKS 2


__global__ 
void testRand ( hiprandState *state, int nb ) {
    int idx = threadIdx.x  + blockIdx.x * blockDim.x;

    printf("Id %i, value %f\n",idx,-5.12 + (hiprand_uniform(&state[idx])) * (5.12 - -5.12));

}
__global__
void setup_kernel (hiprandState* state, unsigned long seed )
{
    int id = threadIdx.x  + blockIdx.x * blockDim.x;
    hiprand_init ( seed, id , 0, &state[id] );
}

/**
* Image comes in in horizontal lines
*/
int main() {
    const dim3 blockSize(MAXTHREADS);
    const dim3 gridSize(NBBLOCKS);

    hiprandState* devStates;
    hipMalloc ( &devStates,MAXTHREADS*NBBLOCKS*sizeof( hiprandState ) );
    time_t t;
    time(&t);
    setup_kernel <<< gridSize, blockSize >>> ( devStates, (unsigned long) t );  
    int nb = 4;
    testRand  <<< gridSize, blockSize >>> ( devStates,nb);  
    testRand  <<< gridSize, blockSize >>> ( devStates,nb);  

    hipFree(devStates);
}