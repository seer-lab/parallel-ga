#include "hip/hip_runtime.h"
#include <stdio.h> 
#include <stdlib.h>
#include <cmath> 
#include <cstring>
#include <vector>
#include <cstdlib>
#include <iostream>
#include <cfloat>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include "utils.h"
#include "timer.h"

using std::vector;
using std::cout;


#define warp 128

void initPopulation(double* population, std::vector<float> bounds, const int row, const int col) {

    for (unsigned int i = 0; i < row; i++)
        for (unsigned int j = 0; j < col; j++)
            *(population + i*col + j) = bounds[0] + ((double)rand() / RAND_MAX) * (bounds[1] - bounds[0]);

}

__device__
double square(double x) { return x * x; }

__global__
void gpu_eval(double *p, double *f, int numRows, int numCols) {

    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    if (tid >= numRows)
        return;

    for (unsigned int i = 0; i < numCols; i++)
        f[tid] += square(p[tid * numCols + i]);

}

void evaluation(double* h_population, double* h_fitness, const int row, const int col, size_t bytesPopulation, size_t bytesFitness) {

    
    double *d_population, *d_fitness;

    hipMalloc(&d_population, bytesPopulation);
    hipMalloc(&d_fitness, bytesFitness);

    hipMemcpy(d_population, h_population, bytesPopulation, hipMemcpyHostToDevice);

    int TB_SIZE = row/warp;

    gpu_eval<<<warp, TB_SIZE>>>(d_population, d_fitness, row, col);

    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

    hipMemcpy(h_fitness, d_fitness, bytesFitness, hipMemcpyDeviceToHost);

    hipFree(d_population);
    hipFree(d_fitness);

}

double square2(double x) { return x * x; }

double sphere(double* individual, int index, const int p) { 

    double sum = 0.0;
    for (std::vector<int>::size_type i = 0; i != p; i++)
        sum += square2(individual[index * p + i]);

    return sum; 
}

void evaluation(double* population, double* fitness, const int populationSize, const int p) {


    for (unsigned int i = 0; i != populationSize; i++) 
        fitness[i] = sphere(population, i, p);

}

void print2dvec(double *v, int r, int c) {

    for (unsigned int i = 0; i < r; i++) {
        cout << "[ ";
        for (unsigned int j = 0; j < c; j++) 
            cout << v[i * c + j] << " ";
        cout << "]\n";
    }

}

void printvec(double *v, int n) {

    for (unsigned int i = 0; i < n; i++)
		cout << v[i] << "\n";
        
}

int bestIndividual(double *a, std::vector<int> b, int n) {
    
    unsigned int i, index = b[0];
    double diff = DBL_MAX;

    for (i = 0; i < n; i++) {
        double absVal = abs(a[b[i]]);
        if (absVal < diff) {
            index = b[i];
            diff = absVal;
        } else if (absVal == diff && a[b[i]] > 0 && a[index] < 0) {
            index = b[i];
        }
    }

    return index;
}

int bestIndividual(double *a, int n) {

    unsigned int i, index = 0;
    double diff = DBL_MAX;
    
    for (i = 0; i < n; i++) {
        double absVal = abs(a[i]);
        if (absVal < diff) {
            index = i;
            diff = absVal;
        } else if (absVal == diff && a[i] > 0 && a[index] < 0) {
            index = i;
        }
    }

    return index;
}


void tournamentSelection(double *parents, double* population, double *fitness, const int p, const int populationSize, const int tournamentSize) {

    std::vector<int> tournamentPool(tournamentSize, 0);

    unsigned int count = 0;
    int parentIndex = 0;
    while (count < populationSize) {
        
        // selecting individuals for tournament
        for (std::vector<int>::size_type i = 0; i != tournamentSize; i++) 
            tournamentPool[i] = rand() % populationSize; 

        parentIndex = bestIndividual(fitness, tournamentPool, tournamentSize);

        
        for (std::vector<int>::size_type i = 0; i != p; i++) 
            parents[count * p + i] = population[parentIndex * p + i];

        count++;
    }

}


void crossover(double *temp_population, double *parents, const int p, const float crossoverProbability, const int mating) {

    int crossoverPoint = 0;


    for (unsigned int i = 0; i < mating; i++) {

        if (((float)rand())/RAND_MAX < crossoverProbability) {
            crossoverPoint = rand() % p;

            for (unsigned int j = 0; j < crossoverPoint; j++) {
                temp_population[i * p + j] = parents[i * p + j];
                temp_population[(i+mating) * p + j] = parents[(i+mating) * p + j];
            }

            for (unsigned int j = crossoverPoint; j < p; j++) {
                temp_population[i * p + j] = parents[(i+mating) * p + j];
                temp_population[(i+mating) * p + j] = parents[i * p + j];
            }

        } else {

            for (unsigned int j = 0; j < p; j++) {
                temp_population[i * p + j] = parents[i * p + j];
                temp_population[(i+mating) * p + j] = parents[(i+mating) * p + j];
            }

        }

    }

}

void mutation(double *temp_population, std::vector<float> bounds, const int p, const int populationSize, const float mutationProbability) {


    for(unsigned int i = 0; i < populationSize; i++) {
        for(unsigned int j = 0; j < p; j++) {
            if (((float)rand())/RAND_MAX < mutationProbability) 
                temp_population[i * p + j] = bounds[0] + ((double)rand() / RAND_MAX) * (bounds[1] - bounds[0]);
        }
    }

}

void replacement (double *population, double* temp_population, double *fitness, const int p, const int populationSize, const int elitism) {

    std::vector<int> minIndex(elitism, 0);


    for(unsigned int i = 0; i < elitism; i++) {
        minIndex[i] = bestIndividual(fitness, populationSize);
        fitness[minIndex[i]] = DBL_MAX;
    }
    
    for(unsigned int i = 0; i < elitism; i++)
        for(unsigned int j = 0; j < p; j++)
            population[i * p + j] = population[minIndex[i] * p + j];

    for(unsigned int i = elitism; i < populationSize; i++)
        for(unsigned int j = 0; j < p; j++)
            population[i * p + j] = temp_population[i * p + j];

}

int main() {

    // Optimization parameters for Sphere function
    vector<float> bounds{-5.12, 5.12};

    // GA parameters
    const int p = 128; // # of genes per individual
    const int populationSize = 32768; 
    const int elitism = 5; 
    const int mating = ceil((populationSize)/2);
    const int tournamentSize = 5;
    int numGenerations = 100; 
    const float crossoverProbability = 0.9f;
    const float mutationProbability = 0.01f;


    // Intialization for random number generator
    time_t t;
    srand((unsigned) time(&t));

    // Vector size
    size_t bytesPopulation = p * populationSize * sizeof(double);
    size_t bytesFitness = populationSize * sizeof(double);

    // Initilize vectors
    double *population, *fitness, *parents, *temp_population, *population_h, *fitness_h, *parents_h, *temp_population_h;

    // Allocate memory
    population = (double*)malloc(bytesPopulation);
    population_h = (double*)malloc(bytesPopulation);
    fitness = (double*)malloc(bytesFitness);
    fitness_h = (double*)malloc(bytesFitness);
    parents = (double*)malloc(bytesPopulation);
    parents_h = (double*)malloc(bytesPopulation);
    temp_population = (double*)malloc(bytesPopulation);
    temp_population_h = (double*)malloc(bytesPopulation);

    //cout << "GPU Part\n";
    // Initialize Population 
    initPopulation(population, bounds, populationSize, p);

    evaluation(population, fitness, populationSize, p, bytesPopulation, bytesFitness);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

    //cout << "GPU Population\n";

    //print2dvec(population, populationSize, p);

    //cout << "GPU Fitness\n";
    // printvec(fitness, populationSize);
    

    // Main GA loop
    while (numGenerations > 0) {

        tournamentSelection(parents, population, fitness, p, populationSize, tournamentSize);
        crossover(temp_population, parents, p, crossoverProbability, mating);
        mutation(temp_population, bounds, p, populationSize, mutationProbability);
        replacement(population, temp_population, fitness, p, populationSize, elitism);
        evaluation(population, fitness, populationSize, p, bytesPopulation, bytesFitness);

        numGenerations--;
    }
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

    //cout << "new fitness GPU" << std::endl;
    //printvec(fitness, populationSize);

    /*
    //cout << "CPU PART \n";
    initPopulation(population_h, bounds, populationSize, p);

    //cout << "CPU Population\n";

    //print2dvec(population_h, populationSize, p);

    numGenerations = 100; 

    evaluation(population_h, fitness_h, populationSize, p);

    //cout << "CPU Fitness\n";
    //printvec(fitness_h, populationSize);

    while (numGenerations > 0) {

        tournamentSelection(parents_h, population_h, fitness_h, p, populationSize, tournamentSize);
        crossover(temp_population_h, parents_h, p, crossoverProbability, mating);
        mutation(temp_population_h, bounds, p, populationSize, mutationProbability);
        replacement(population_h, temp_population_h, fitness_h, p, populationSize, elitism);
        evaluation(population_h, fitness_h, populationSize, p);

        numGenerations--;
    }

    //cout << "new fitness CPU" << std::endl;
    //printvec(fitness_h, populationSize);
    */
    return 0;
}
