#include "hip/hip_runtime.h"
#include <stdio.h> 
#include <stdlib.h>
#include <cstring>
#include <cstdlib>
#include <math.h>
#include <algorithm>

#include "include/utils.h" // For cuda imports

#include "include/migration.cuh"
#include "include/mutation.cuh"
#include "include/crossover.h"
#include "include/selection.cuh"
#include "include/evaluation.cuh"
#include "include/population.h"
#include "../constants.h"

using std::cout;
using std::endl;

__global__
void setup_kernel (hiprandState* state, unsigned long seed) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init (seed, tid , 0, &state[tid]);
}

__global__
void gpu_GA_pre_crossover(hiprandState *d_state, double* population, double* fitness, double* parents, const int populationSize, const int p, int numGenerations) { 

    // Thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (numGenerations == numGen) {
        if (evaluation_type == 1) 
            sphere(fitness, population, p, tid);
        else if (evaluation_type == 2)
            rastrigin(fitness, population, p, tid);
        else if (evaluation_type == 3)
            ackley(fitness, population, p, tid);
        else
            griewank(fitness, population, p, tid);
        __syncthreads();
    }
    
    // Tournament Selection
    selection(d_state, parents, population, fitness, p, tid, individualsPerIsland);
    __syncthreads();
}   

__global__
void gpu_GA_post_crossover(hiprandState *d_state, double* population, double* fitness, double* parents, const int populationSize, const int p, int numGenerations) {

        // Thread ID
        int tid = blockIdx.x * blockDim.x + threadIdx.x;

        // Every 5 generations migrate individuals to n+1 islands
        bool migrationFlag = false;
        if (numGenerations % 2 == 0) migrationFlag = true;

        // TODO: Guassian Mutation (Look into implementing guassian mutation)
        mutation(d_state, population, p, tid, lowerBound, upperBound, mutationProbability);
        __syncthreads();

        // TODO: Migration
        if (migrationFlag)
            migration(population, fitness, p, tid, elitism, individualsPerIsland, islands);
        __syncthreads();

        fitness[tid] = 0;
        __syncthreads();

        // Evaluation for each individual
        if (evaluation_type == 1) 
            sphere(fitness, population, p, tid);
        else if (evaluation_type == 2)
            rastrigin(fitness, population, p, tid);
        else if (evaluation_type == 3)
            ackley(fitness, population, p, tid);
        else
            griewank(fitness, population, p, tid);
        __syncthreads();
}

void parallelGA(double* h_population, 
                double* h_fitness, 
                const int populationSize, 
                const int p, 
                size_t bytesPopulation, 
                size_t bytesFitness,
                int numGenerations,
                hiprandState *&d_state,
                const int mating,
                time_t t) {
    
    // Allocating device memory
    double *d_population, *d_fitness, *d_parents, *h_parents;

    h_parents = (double*)malloc(bytesPopulation);

    hipMalloc(&d_population, bytesPopulation);
    hipMalloc(&d_fitness, bytesFitness);
    hipMalloc(&d_parents, bytesPopulation);

    hipMemset(d_fitness, 0, bytesFitness);
    hipMemset(d_parents, 0, bytesPopulation);
    
    // Copying population to device (intend to remove in the future)
    hipMemcpy(d_population, h_population, bytesPopulation, hipMemcpyHostToDevice);
    
    while (numGenerations > 0) {

        setup_kernel<<<islands,individualsPerIsland>>>(d_state, (unsigned long) t );
        hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());  

        gpu_GA_pre_crossover<<<islands, 
                 individualsPerIsland>>>
                 (d_state, d_population, d_fitness, d_parents, populationSize, p, numGenerations);
        hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());  
        
        // Copy population and fitness back to host
        hipMemcpy(h_population, d_population, bytesPopulation, hipMemcpyDeviceToHost);
        hipMemcpy(h_parents, d_parents, bytesPopulation, hipMemcpyDeviceToHost);

        // CPU Crossover
        if (crossover_type == 1) {
            arithmetic_crossover(h_population, h_parents, p, crossoverProbability, mating, alpha);
        } else if (crossover_type == 2) {
            simulated_binary_crossover(h_population, h_parents, p, crossoverProbability, mating, nc);
        } else {
            line_crossover(h_population, h_parents, p, crossoverProbability, mating);
        }

        // Copying population to device (intend to remove in the future)
        hipMemcpy(d_population, h_population, bytesPopulation, hipMemcpyHostToDevice);
        hipMemcpy(d_parents, h_parents, bytesPopulation, hipMemcpyHostToDevice);

        gpu_GA_post_crossover<<<islands, 
                 individualsPerIsland>>>
                 (d_state, d_population, d_fitness, d_parents, populationSize, p, numGenerations);
        hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());  
        numGenerations--;
    }

    // Copy population and fitness back to host
    hipMemcpy(h_population, d_population, bytesPopulation, hipMemcpyDeviceToHost);
    hipMemcpy(h_fitness, d_fitness, bytesFitness, hipMemcpyDeviceToHost);

    hipFree(d_population);
    hipFree(d_fitness);
    hipFree(d_state);

}

void print2dvec(double *v, int r, int c) {
    for (unsigned int i = 0; i < r; i++) {
        cout << "[ ";
        for (unsigned int j = 0; j < c; j++) 
            cout << v[i * c + j] << " ";
        cout << "]\n";
    }
}

void printvec(double *v, int n) {
    for (unsigned int i = 0; i < n; i++)
		cout << v[i] << "\n";
}


int main() {

    // Optimization parameters for Sphere function
    float bounds[2] = {lowerBound, upperBound};

    // GA parameters
    const int p = p1; // # of genes per individual
    const int populationSize = populationSize1; 
    const int mating = mating1;
    int numGenerations = numGen; 

    // Intialization for random number generator
    time_t t;
    srand((unsigned) time(&t));

    // Vector size
    size_t bytesPopulation = p * populationSize * sizeof(double);
    size_t bytesFitness = populationSize * sizeof(double);

    // Initilize vectors
    double *population, *fitness;

    // Allocate memory
    population = (double*)malloc(bytesPopulation);
    fitness = (double*)malloc(bytesFitness);

    // Initialize Population 
    initPopulation(population, bounds, populationSize, p);

    // cuRand setup
    hiprandState *d_state;
    hipMalloc(&d_state, islands*individualsPerIsland*sizeof( hiprandState ) );

    // GA
    parallelGA(population, fitness, populationSize, p, bytesPopulation, bytesFitness, numGenerations, d_state, mating, t);

    // printvec(fitness, populationSize);

    double *min = std::min_element(fitness, fitness + populationSize);

    // Find the minimum element
    cout << "\nMin Element = " << *min << "\t" << crossover_type << endl;

    return 0;
}