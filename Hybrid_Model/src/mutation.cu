#include "../include/mutation.cuh"

__device__
void mutation(hiprandState *d_state, double *population, const int p, int tid, 
              const float lowerBound, const float upperBound, const float mutationProbability) {

    for (unsigned int i = 0; i < p; i++) {
        float myrand = hiprand_uniform(&d_state[tid]);
        if (myrand < mutationProbability) 
            population[tid * p + i] = lowerBound + (hiprand_uniform(&d_state[tid])) * (upperBound - lowerBound);
    }
}