#include "hip/hip_runtime.h"
#include "evaluation.cuh"

__device__
double square(double x) { return x * x; }

__global__
void sphere_eval(double *p, double *f, int numRows, int numCols) {
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    // Return if we're out of bounds
    if (tid >= numRows)
        return;

    f[tid] = 0;
    __syncthreads();
    
    for (unsigned int i = 0; i < numCols; i++)
        f[tid] += square(p[tid * numCols + i]);

}

void evaluation(int warp, double* h_population, double* h_fitness, double *d_population, double *d_fitness, const int row, const int col, size_t bytesPopulation, size_t bytesFitness) {

    // Copying memory onto device
    hipMemcpy(d_population, h_population, bytesPopulation, hipMemcpyHostToDevice);

    // Threads per block
    int TPB_SIZE = row/warp;

    sphere_eval<<<warp, TPB_SIZE>>>(d_population, d_fitness, row, col);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

    // Copy back to host
    hipMemcpy(h_fitness, d_fitness, bytesFitness, hipMemcpyDeviceToHost);

}